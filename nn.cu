
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include <time.h>

#define OUTPUT 2 //taille output
#define BATCH_SIZE 2
#define LEARNING_RATE 0.1f

struct layer {
    int n; //taille layer sortie a
    int p; // taille layer entrée x
    float* w; // dim (n, p)
    float* x; // dim (p, 1) * BATCH_size
    float* b; // dim (n, 1)
    float* a; // dim (n, 1) * BATCH_size
    float* z; //dim (n, 1) * BATCH_size
    float* wT; //dim (p, n)
    float* aT; //dim (1, n) * BATCH_size
    float* da; //dim (n, 1) * BATCH_size
    float* dw; //dim (n, p) * BATCH_size
} typedef layer;

struct network {
    int nb_layers;
    layer** layers;
    float* y;
    float* error;
} typedef network;

float normal_distribution(float mean, float stddev) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float z0 = sqrt(-2.0 * log(u1)) * cos(2.0 * M_PI * u2);
    return z0 * stddev + mean;
}

// Initialisation des poids selon la méthode de He
void initialize_weights_he(float* w, int p, int n) {
    // Calcul de l'écart type (stddev)
    float stddev = sqrt(2.0 / p);

    // Remplir les poids avec des valeurs suivant la distribution normale
    for (int i=0; i<n; i++) {
        for(int j=0; j<p; j++) {
            w[i*p + j] = normal_distribution(0.0, stddev);
        }
    }
}

void read_csv(float* x, float* y, int* number_columns, int* number_observations, char* file_name) {
    FILE *file;
    char *buffer;
    long file_size;

    // Ouvrir le fichier en mode binaire
    file = fopen(file_name, "rb");
    if (file == NULL) {
        perror("Erreur lors de l'ouverture du fichier");
        exit(EXIT_FAILURE);
    }

    // Se positionner à la fin du fichier pour déterminer sa taille
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file); // Revenir au début du fichier

    // Allouer la mémoire pour stocker le contenu du fichier
    buffer = (char *)malloc(sizeof(char) * file_size + 1);
    if (buffer == NULL) {
        perror("Erreur d'allocation de mémoire");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    // Lire tout le fichier dans le buffer
    fread(buffer, sizeof(char), file_size, file);
    buffer[file_size] = '\0';

    //calcul nombre de colonnes et nombre de lignes
    *number_columns = 0;
    *number_observations = 0;

    int c;
    bool first_row = true;

    while((c = fgetc(file)) != EOF) {
        if(first_row && c == ',') {
            *number_columns += 1;
        }
        
        if(c == '\n') {
            *number_observations += 1;
        }
        
    }

    x = (float*)malloc(sizeof(float)*(*number_columns)*(*number_observations));
    y = (float*)malloc(sizeof(float)*(*number_observations)*3);

    int nb_char_per_line = (*number_columns)*30;

    char* line = (char*)malloc(sizeof(char)*nb_char_per_line); //chaque colonne ne dépasse pas 30 caractères
    char * strToken;
    int count;
    int line_count = 0;
    char *endptr;

    while(fgets(line, nb_char_per_line, file) != NULL) {
        count = 0;
        strToken = strtok (line, ",");

        while ( strToken != NULL ) {
            if(count < (*number_columns)) {
                x[count + line_count*(*number_columns)] = strtof(strToken, &endptr);
            }
            else{
                float num = strtof(strToken, &endptr);
                for(int i=0; i<3; i++) {
                    y[line_count*3 + i] = (num == (float)i) * i;
                }
            }
            strToken = strtok ( NULL, "," );
        }

        line_count++;
    }




    fclose(file);
    free(buffer);
}

float random_float_0_to_1() {
    return (float)rand() / (float)RAND_MAX;
}

void handle_malloc(void** dp, int size) {
    if(hipMalloc(dp, size) > 0) {
        printf("Malloc error for pointeur of size %d\n", size);
        exit(EXIT_FAILURE);
    }
}

void handle_copy_of_network(network* net_to_copy, network* other, int direction) {
    
}

//initialise les connexions entre deux layers
layer* create_layer(int p, int n) {

    if(p > 1024 || n > 1024) {
        printf("Layers with dimensionality above 1024 are prohibited\n");
        exit(EXIT_FAILURE);
    }
    layer* l = (layer*)malloc(sizeof(layer));

    l->n = n;
    l->p = p;

    l->w = (float*)malloc(sizeof(float)*n*p); 
    l->x = NULL; 
    l->b = (float*)malloc(sizeof(float)*n);
    l->a = (float*)malloc(sizeof(float)*n*BATCH_SIZE);
    l->z = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    l->wT = (float*)malloc(sizeof(float)*n*p*BATCH_SIZE);
    l->aT = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    //multiplier par batch size
    l->dw = (float*)malloc(sizeof(float)*n*p*BATCH_SIZE);
    l->da = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    initialize_weights_he(l->w, l->p, l->n);

    for(int i=0; i<l->n; i++) {
        l->b[i] = 0;
    }

    return l;
}



network* create_network_with_layers(int nb_layers, ...) {
    va_list args;

    network* n = (network*)malloc(sizeof(network));
    n->nb_layers = nb_layers;
    n->layers = (layer**)malloc(sizeof(layer*)*nb_layers);

    va_start(args, nb_layers);

    for(int i=0; i<nb_layers; i++) {
        n->layers[i] = va_arg(args, layer*);
    }

    va_end(args);

    return n;
}

network* create_empty_network() {
    network* n = (network*)malloc(sizeof(network));
    n->layers = NULL;
    n->nb_layers = 0;
    n->error = (float*)malloc(sizeof(float));
    n->y = NULL;

    return n;
}

void load_new_batch(float* x, float* y, network* net) {
    if(net->nb_layers == 0) {
        printf("Empty network\n");
        exit(EXIT_FAILURE);
    }

    net->y = y;

    net->layers[0]->x = x;

    net->error[0] = 0;
}

void add_layer_to_network(network* n, layer* l) {
    if(n->nb_layers > 0) {
        if(n->layers[n->nb_layers-1]->n != l->p) {
            printf("Incompatible size for layers : size n for layer (i) must be equal t osize p for layer (i+1)\n");
            exit(EXIT_FAILURE);
        }
    }

    n->nb_layers += 1;

    if(n->layers == NULL) {
        n->layers = (layer**)malloc(sizeof(layer*));
    }
    else{
        n->layers = (layer**)realloc(n->layers, sizeof(layer*)*(n->nb_layers));
    }

    n->layers[n->nb_layers-1] = l;
}

void cross_entropy(network* net) {
    net->error[0] = 0;
    for(int i=0; i<net->layers[net->nb_layers-1]->n*BATCH_SIZE; i++) {
        net->error[0] += net->y[i] * logf(net->layers[net->nb_layers-1]->a[i]);
    }
    net->error[0] *= -1;
}

__device__ void k_transpose_w(float* w, float* wT, int n, int p) {
    int lid = threadIdx.x;
    int begin_batch = blockIdx.x * n * p;

    if(lid < n) {
        for(int j=0; j<p; j++) {
            wT[j*n + begin_batch + lid] = w[begin_batch + lid*p + j];
        }
    }
}

//calcule le gradient pour la dernière couche 
__device__ void k_gradient_last_layer(float* da, float* a, float* y, int n) {
    int lid = threadIdx.x;
    int ind = blockIdx.x*n + lid;

    if(lid < n) {
        da[ind] = a[ind] - y[ind];
    }
}

//calcule : grad_l {p_l+1/n_l, 1} = (w_l+1)^T{p_l+1, n_l+1} * grad_l+1 {n_l+1, 1} * deriv(ReLu(z_l) {n_l, 1})
// p = p_l+1 et n = n_l+1
__device__ void k_gradient_hidden_layer(float* da, float* da_next, float* w, float* z, int p, int n) {
    int lid = threadIdx.x;

    __shared__ float sh_da_next[1024];

    int begin_batch = blockIdx.x*p;
    float res = 0;

    if(lid < n) {
        sh_da_next[lid] = da_next[lid + blockIdx.x*n];
    }
    __syncthreads();


    if (lid < p){

        for(int row=0; row<n; row++) {
            res += w[lid*n + row] * sh_da_next[row];
        }

        da[lid + begin_batch] = res * (z[lid + begin_batch] > 0);

    }
}

//dw {n_l, p_l} = (grad_l) {n_l, 1} * (a_l-1) {1, p_l}
__device__ void k_gradient_w(float* dw, float* da, float* a_previous, int n, int p){
    int lid = threadIdx.x;

    __shared__ float shda[1024];
    __shared__ float sha[1024];

    int begin_batch_w = blockIdx.x * n * p;
    int begin_batch_da = blockIdx.x * n;
    int begin_batch_a = blockIdx.x * p;

    if(lid < n) {
        shda[lid] = da[begin_batch_da + lid];
    }

    if(lid < p) {
        sha[lid] = a_previous[begin_batch_a + lid];
    }

    __syncthreads();

    if(lid < n) {
        for(int col=0; col<p; col++) {
            dw[begin_batch_w + lid*n + col] = shda[lid] * sha[col];
        }
    }
}

// w_l {n_l, p_l} = w_l {n_l, p_l} - lr * dw {n_l, p_l}
__device__ void k_update_weights(float* w, float* dw, int n, int p, float learn_rate) {
    int lid = threadIdx.x;
    int begin_batch = blockIdx.x * n * p;

    if(n > p) {
        if(lid < n) {
            for(int col=0; col<p; col++) {
                atomicAdd(&w[lid*n + col],  -learn_rate * w[begin_batch + lid*n + col]/BATCH_SIZE);
            }
        }
    }
    else {
        if(lid < p) {
            for(int row=0; row<n; row++) {
                atomicAdd(&w[lid*p + row], -learn_rate * w[begin_batch + lid*p + row]/BATCH_SIZE);
            }
        }
    }
}

// b_l {n_l, 1} = b_l {n_l, 1} - lr * da{n_l, 1}
__device__ void k_update_bias(float* b, float* da, int n, float learn_rate) {
    int lid = threadIdx.x;
    int begin_batch = blockIdx.x * n;

    if (lid < n){    
        atomicAdd(&b[lid], -learn_rate * da[begin_batch + lid]/BATCH_SIZE);
    }
}


__global__ void k_back_propagation(network* net) {
    layer* l;
    for(int i=net->nb_layers-1; i>-1;i--) {
        l = net->layers[i];
        if(i == net->nb_layers-1) {
            k_gradient_last_layer(l->da, l->a, net->y, l->n);
        }
        else {
            k_transpose_w(net->layers[i+1]->w, net->layers[i+1]->wT, net->layers[i+1]->n, net->layers[i+1]->p);

            __syncthreads();

            k_gradient_hidden_layer(l->da, net->layers[i+1]->da, net->layers[i+1]->wT, l->z, net->layers[i+1]->p, net->layers[i+1]->n);
        }

        __syncthreads();

        if(i > 0) {
            k_gradient_w(l->dw, l->da, net->layers[i-1]->a, l->n, l->p);
        }
        else {
            k_gradient_w(l->dw, l->da, l->x, l->n, l->p);
        }

        __syncthreads();

        k_update_weights(l->w, l->dw, l->n, l->p, LEARNING_RATE);

        __syncthreads();

        k_update_bias(l->b, l->da, l->n, LEARNING_RATE);

        __syncthreads();
    }
}

// result{n, 1} = o(w_{n, p} * x_{p, 1} + b_{n, 1})
__device__ void k_feed_forward(float* x, float* w, float* bias, float* z, float* a, int n, int p, bool last_layer) {
    int lid = threadIdx.x;

    __shared__ float shx[1024];

    int begin_batch = blockIdx.x*n;
    float res = 0;

    if(lid < p) {
        shx[lid] = x[lid + blockIdx.x*p];
    }
    __syncthreads();


    if (lid < n){


        for(int col=0; col<p; col++) {
            res += w[lid*p + col] * shx[col];
        }

        res += bias[lid + begin_batch];

        z[lid + begin_batch] = res;

    }

    __syncthreads();

    if(lid < n) {
        //activation softmax pour le dernier layer
        if(last_layer) {
            float sum = 0;
            for(int j=0; j<n; j++) {
                sum += expf(z[j + begin_batch]);
            }
            a[lid + begin_batch] = expf(z[lid + begin_batch])/sum; 
        }
        //activation ReLu pour les autres
        else {
            a[lid + begin_batch] = z[lid + begin_batch] * (z[lid + begin_batch] >= 0);
        }
    }
}

__global__ void k_step(network* n) {
    bool last_layer;
    for(int i=0; i<n->nb_layers; i++) {
        layer* l = n->layers[i];

        last_layer = i == (n->nb_layers-1);

        if (i == 0) {
            k_feed_forward(l->x, l->w, l->b, l->z, l->a, l->n, l->p, last_layer);
        }
        else
        {
            k_feed_forward(n->layers[i-1]->a, l->w, l->b, l->z, l->a, l->n, l->p, last_layer);
        }
    } 
}

int main(int argc, char **argv){

    network* net = create_empty_network();

    add_layer_to_network(net, create_layer(2, 4));
    add_layer_to_network(net, create_layer(4, 2));

    float x[4] = {1, 1, 1, 1};
    float y[2*BATCH_SIZE] = {1.0f, 0, 1.0f, 0}; //nombre de classes * nombre de d'obervastions

    load_new_batch(x, y, net);

    network *dnet;
    handle_malloc((void**)&dnet, sizeof(network));

    hipMemcpy(dnet, net, sizeof(network), hipMemcpyHostToDevice);

    printf("b² before update: \n");
    for(int i=0; i<net->layers[1]->n; i++){
        printf("%f\n", net->layers[1]->b[i]);
    }

    printf("W² before update: \n");
    for(int i=0; i<net->layers[1]->n*net->layers[1]->p; i++){
        printf("%f\n", net->layers[1]->w[i]);
    } 

    k_step<<<BATCH_SIZE, 1024>>>(dnet);

    k_back_propagation<<<BATCH_SIZE, 1024>>>(dnet);

    hipMemcpy(net, dnet, sizeof(network), hipMemcpyDeviceToHost);

    printf("X : \n");
    for(int i=0; i<net->layers[0]->p*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[0]->x[i]);
    }

    printf("W¹ : \n");
    for(int i=0; i<net->layers[0]->n*net->layers[0]->p; i++){
        if(i%net->layers[0]->p == 0) {
            printf("----------%d\n", i/net->layers[0]->p);
        }
        printf("%f\n", net->layers[0]->w[i]);
    }

    printf("Z¹ : \n");
    for(int i=0; i<net->layers[0]->n*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[0]->z[i]);
    }

    printf("A¹ : \n");
    for(int i=0; i<net->layers[0]->n*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[0]->a[i]);
    }

    printf("b² after update: %d\n", net->layers[1]->n);
    for(int i=0; i<net->layers[1]->n; i++){
        printf("%f\n", net->layers[1]->b[i]);
    }

    printf("W² after update: \n");
    for(int i=0; i<net->layers[1]->n*net->layers[1]->p; i++){
        if(i%net->layers[1]->p == 0) {
            printf("----------%d\n", i/net->layers[1]->p);
        }
        printf("%f\n", net->layers[1]->w[i]);
    }

    printf("Z² : \n");
    for(int i=0; i<net->layers[1]->n*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[1]->z[i]);
    }

    printf("results : \n");
    for(int i=0; i<net->layers[net->nb_layers-1]->n*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[net->nb_layers-1]->a[i]);
    }

    printf("WT : \n");
    for(int i=0; i<net->layers[1]->n*net->layers[0]->p; i++){
        printf("%f\n", net->layers[1]->wT[i]);
    }

    printf("Error : %f\n", net->error[0]);

    cross_entropy(net);

    printf("Error : %f\n", net->error[0]);
    
    hipFree(dnet);

    return 0;
}