
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include <time.h>

#define N 50 //taille layer sortie
#define P 70 //taille layer entrée
#define OUTPUT 2 //taille output
#define BATCH_SIZE 2
#define LEARNING_RATE 0.1f

struct layer {
    int n; //taille layer sortie a
    int p; // taille layer entrée x
    float* w; // dim (n, p)
    float* x; // dim (p, 1) * BATCH_size
    float* b; // dim (n, 1)
    float* a; // dim (n, 1) * BATCH_size
    float* z; //dim (n, 1) * BATCH_size
    float* wT; //dim (p, n)
    float* aT; //dim (1, n) * BATCH_size
    float* da; //dim (n, 1) * BATCH_size
    float* dw; //dim (n, p) * BATCH_size
} typedef layer;

struct network {
    int nb_layers;
    layer** layers;
    float* y;
    float* error;
} typedef network;

float normal_distribution(float mean, float stddev) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float z0 = sqrt(-2.0 * log(u1)) * cos(2.0 * M_PI * u2);
    return z0 * stddev + mean;
}

// Initialisation des poids selon la méthode de He
void initialize_weights_he(float* w, int p, int n) {
    // Calcul de l'écart type (stddev)
    float stddev = sqrt(2.0 / p);

    // Remplir les poids avec des valeurs suivant la distribution normale
    for (int i=0; i<n; i++) {
        for(int j=0; j<p; j++) {
            w[i*p + j] = normal_distribution(0.0, stddev);
        }
    }
}

void read_csv(float* x, float* y, int* number_columns, int* number_observations, char* file_name) {
    FILE *file;
    char *buffer;
    long file_size;

    // Ouvrir le fichier en mode binaire
    file = fopen(file_name, "rb");
    if (file == NULL) {
        perror("Erreur lors de l'ouverture du fichier");
        exit(EXIT_FAILURE);
    }

    // Se positionner à la fin du fichier pour déterminer sa taille
    fseek(file, 0, SEEK_END);
    file_size = ftell(file);
    rewind(file); // Revenir au début du fichier

    // Allouer la mémoire pour stocker le contenu du fichier
    buffer = (char *)malloc(sizeof(char) * file_size + 1);
    if (buffer == NULL) {
        perror("Erreur d'allocation de mémoire");
        fclose(file);
        exit(EXIT_FAILURE);
    }

    // Lire tout le fichier dans le buffer
    fread(buffer, sizeof(char), file_size, file);
    buffer[file_size] = '\0';

    //calcul nombre de colonnes et nombre de lignes
    *number_columns = 0;
    *number_observations = 0;

    int c;
    bool first_row = true;

    while((c = fgetc(file)) != EOF) {
        if(first_row && c == ',') {
            *number_columns += 1;
        }
        
        if(c == '\n') {
            *number_observations += 1;
        }
        
    }

    x = (float*)malloc(sizeof(float)*(*number_columns)*(*number_observations));
    y = (float*)malloc(sizeof(float)*(*number_observations)*3);

    int nb_char_per_line = (*number_columns)*30;

    char* line = (char*)malloc(sizeof(char)*nb_char_per_line); //chaque colonne ne dépasse pas 30 caractères
    char * strToken;
    int count;
    int line_count = 0;
    char *endptr;

    while(fgets(line, nb_char_per_line, file) != NULL) {
        count = 0;
        strToken = strtok (line, ",");

        while ( strToken != NULL ) {
            if(count < (*number_columns)) {
                x[count + line_count*(*number_columns)] = strtof(strToken, &endptr);
            }
            else{
                float num = strtof(strToken, &endptr);
                for(int i=0; i<3; i++) {
                    y[line_count*3 + i] = (num == (float)i) * i;
                }
            }
            strToken = strtok ( NULL, "," );
        }

        line_count++;
    }




    fclose(file);
    free(buffer);
}

float random_float_0_to_1() {
    return (float)rand() / (float)RAND_MAX;
}

void handle_malloc(void** dp, int size) {
    if(hipMalloc(dp, size) > 0) {
        printf("Malloc error for pointeur of size %d\n", size);
        exit(EXIT_FAILURE);
    }
}

//initialise les connexions entre deux layers
layer* create_layer(int p, int n) {
    layer* l = (layer*)malloc(sizeof(layer));

    l->n = n;
    l->p = p;

    l->w = (float*)malloc(sizeof(float)*n*p); 
    l->x = NULL; 
    l->b = (float*)malloc(sizeof(float)*n);
    l->a = (float*)malloc(sizeof(float)*n*BATCH_SIZE);
    l->z = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    l->wT = (float*)malloc(sizeof(float)*n*p*BATCH_SIZE);
    l->aT = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    //multiplier par batch size
    l->dw = (float*)malloc(sizeof(float)*n*p*BATCH_SIZE);
    l->da = (float*)malloc(sizeof(float)*n*BATCH_SIZE);

    initialize_weights_he(l->w, l->p, l->n);

    for(int i=0; i<l->n; i++) {
        l->b[i] = 0;
    }

    return l;
}



network* create_network_with_layers(int nb_layers, ...) {
    va_list args;

    network* n = (network*)malloc(sizeof(network));
    n->nb_layers = nb_layers;
    n->layers = (layer**)malloc(sizeof(layer*)*nb_layers);

    va_start(args, nb_layers);

    for(int i=0; i<nb_layers; i++) {
        n->layers[i] = va_arg(args, layer*);
    }

    va_end(args);

    return n;
}

network* create_empty_network() {
    network* n = (network*)malloc(sizeof(network));
    n->layers = NULL;
    n->nb_layers = 0;
    n->error = (float*)malloc(sizeof(float));
    n->y = NULL;

    return n;
}

void load_new_batch(float* x, float* y, network* net) {
    if(net->nb_layers == 0) {
        printf("Empty network\n");
        exit(EXIT_FAILURE);
    }

    net->y = y;

    net->layers[0]->x = x;

    net->error[0] = 0;
}

void add_layer_to_network(network* n, layer* l) {
    if(n->nb_layers > 0) {
        if(n->layers[n->nb_layers-1]->n != l->p) {
            printf("Incompatible size for layers : size n for layer (i) must be equal t osize p for layer (i+1)\n");
            exit(EXIT_FAILURE);
        }
    }

    n->nb_layers += 1;

    if(n->layers == NULL) {
        n->layers = (layer**)malloc(sizeof(layer*));
    }
    else{
        n->layers = (layer**)realloc(n->layers, sizeof(layer*)*(n->nb_layers));
    }

    n->layers[n->nb_layers-1] = l;
}

void cross_entropy(network* net) {
    for(int i=0; i<net->layers[net->nb_layers-1]->n*BATCH_SIZE; i++) {
        net->error[0] += net->y[i] * logf(net->layers[net->nb_layers-1]->a[i]);
    }
    net->error[0] *= -1;
}

__device__ void k_transpose_w(float* w, float* wT, int n, int p) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if(thread_number < n) {
        int row = thread_number%n;
        for(int j=0; j<p; j++) {
            wT[j * n + row] = w[row * p + j];
        }
    }
}

//calcule le gradient pour la dernière couche 
__device__ void k_gradient_last_layer(float* da, float* a, float* y, int n) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if(thread_number < n*BATCH_SIZE) {
        da[thread_number] = a[thread_number] - y[thread_number];
    }
}

//calcule : grad_l {p_l+1/n_l, 1} = (w_l+1)^T{p_l+1, n_l+1} * grad_l+1 {n_l+1, 1} * deriv(ReLu(z_l) {n_l, 1})
// p = p_l+1 et n = n_l+1
__device__ void k_gradient_hidden_layer(float* da, float* da_next, float* w, float* z, int p, int n) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < p*BATCH_SIZE){
        int row = thread_number%p;
        float res = 0;
        int begin = thread_number/p * n;
        for(int i=0;i<n;i++){
            res += w[row * n + i] * da_next[i + begin];
        }
        da[thread_number] = res * (z[thread_number] > 0);
        
    }
}

__device__ void k_gradient_w(float* dw, float* da, float* a_previous, int n, int p){
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < n*BATCH_SIZE){
        int row = thread_number%n;
        int begin = thread_number - thread_number%(n*p);
        for(int i=0;i<p;i++){
            dw[row * p + i + begin] = da[thread_number/BATCH_SIZE * n + i] * a_previous[thread_number/BATCH_SIZE * p + i];
        }       
    }
}

__device__ void k_update_weights(float* w, float* dw, int n, int p, float learn_rate) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < n*p){
        float total = 0;
        for(int i=0;i<BATCH_SIZE;i++){
            total += dw[thread_number%(n*p) + (i*n*p)];
        }       
        w[thread_number] -= learn_rate * (total/BATCH_SIZE);
    }
}

__device__ void k_update_bias(float* b, float* da, int n, float learn_rate) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < n){
        float total = 0;
        for(int i=0;i<BATCH_SIZE;i++){
            total += da[thread_number%n + (i*n)];
        }       
        b[thread_number] -= learn_rate * (total/BATCH_SIZE);
    }
}

__device__ void k_back_propagation(network* net) {
    layer* l;
    for(int i=net->nb_layers-1; i>-1;i--) {
        l = net->layers[i];
        if(i == net->nb_layers-1) {
            k_gradient_last_layer(l->da, l->a, net->y, l->n);
        }
        else {
            k_transpose_w(net->layers[i+1]->w, net->layers[i+1]->wT, net->layers[i+1]->n, net->layers[i+1]->p);

            __syncthreads();

            k_gradient_hidden_layer(l->da, net->layers[i+1]->da, net->layers[i+1]->wT, l->z, net->layers[i+1]->p, net->layers[i+1]->n);
        }

        __syncthreads();

        if(i > 0) {
            k_gradient_w(l->dw, l->da, net->layers[i-1]->a, l->n, l->p);
        }
        else {
            k_gradient_w(l->dw, l->da, l->x, l->n, l->p);
        }

        __syncthreads();

        k_update_weights(l->w, l->dw, l->n, l->p, LEARNING_RATE);

        __syncthreads();

        k_update_bias(l->b, l->da, l->n, LEARNING_RATE);

        __syncthreads();
    }
}

// result{n, 1} = w_{n, p} * a_{p, 1} + b_{n, 1}
__device__ void k_feed_forward(float* x, float* w, float* bias, float* z, int n, int p) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < n*BATCH_SIZE){
        int row = thread_number%n;
        float res = 0;
        int begin_x = thread_number/n * p;
        for(int i=0;i<p;i++){
            res += w[row * p + i] * x[i + begin_x];
        }
        z[thread_number] = res + bias[row];
    }
}

__device__ void k_activation(float* z, float* a, int p, int n, bool last_layer) {
    int thread_number = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_number < n*BATCH_SIZE){
    //activation softmax pour le dernier layer
        if(last_layer) {
            float sum = 0;
            int begin = thread_number - (thread_number%n);
            for(int j=0; j<n; j++) {  //commun pour les n valeurs de result
                sum += expf(z[j + begin]);
            }

            a[thread_number] = expf(z[thread_number])/sum; 
        }
        //activation ReLu pour les autres
        else {
            a[thread_number] = z[thread_number] * (z[thread_number] >= 0);
        }
    }
}

__global__ void k_step(network* n) {
    bool last_layer;
    for(int i=0; i<n->nb_layers; i++) {
        layer* l = n->layers[i];

        last_layer = i == (n->nb_layers-1);

        if (i == 0) {
            k_feed_forward(l->x, l->w, l->b, l->z, l->n, l->p);
        }
        else
        {
            k_feed_forward(n->layers[i-1]->a, l->w, l->b, l->z, l->n, l->p);
        }
        __syncthreads();

        k_activation(l->z, l->a, l->p, l->n, last_layer);

        __syncthreads();
    } 

    /* k_back_propagation(n);

    __syncthreads(); */
}

int main(int argc, char **argv){

    network* net = create_empty_network();

    add_layer_to_network(net, create_layer(4, 30));
    add_layer_to_network(net, create_layer(30, 3));

    float x[8] = {0.6530f, 0.2698f, 0.7625f, 0.9401f, 0.48f, 0.59f, 0.85f, 0.95f};
    float y[3*BATCH_SIZE] = {1.0f, 0, 1.0f, 0, 1.0, 0}; //nombre de classes * nombre de d'obervastions

    load_new_batch(x, y, net);

    network *dnet;
    handle_malloc((void**)&dnet, sizeof(network));

    hipMemcpy(dnet, net, sizeof(network), hipMemcpyHostToDevice);

    printf("W² before update: \n");
    for(int i=0; i<net->layers[1]->n*net->layers[1]->p; i++){
        printf("%f\n", net->layers[1]->w[i]);
    }

    for(int i=0; i<50; i++){
        k_step<<<1, 1024>>>(net);
    }

    hipMemcpy(net, dnet, sizeof(network), hipMemcpyDeviceToHost);

    printf("X : \n");
    for(int i=0; i<net->layers[0]->p*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[0]->x[i]);
    }

    printf("W¹ : \n");
    for(int i=0; i<net->layers[0]->n*net->layers[0]->p; i++){
        printf("%f\n", net->layers[0]->w[i]);
    }

    printf("A¹ : \n");
    for(int i=0; i<net->layers[0]->n*BATCH_SIZE; i++){
        if(i%2 == 0) {
            printf("----------%d\n", i/2);
        }
        printf("%f\n", net->layers[0]->a[i]);
    }

    printf("W² after update: \n");
    for(int i=0; i<net->layers[1]->n*net->layers[1]->p; i++){
        printf("%f\n", net->layers[1]->w[i]);
    }

    printf("results : \n");
    for(int i=0; i<net->layers[net->nb_layers-1]->n*BATCH_SIZE; i++){
        if(i%3 == 0) {
            printf("----------%d\n", i/3);
        }
        printf("%f\n", net->layers[net->nb_layers-1]->a[i]);
    }

    printf("WT : \n");
    for(int i=0; i<net->layers[1]->n*net->layers[0]->p; i++){
        printf("%f\n", net->layers[1]->wT[i]);
    }

    cross_entropy(net);

    printf("Error : %f\n", net->error[0]);
    
    hipFree(dnet);

    return 0;
}